#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#if CUDA_VERSION >= 10010

#include <cstring>
#include <vector>
#include <hip/hip_runtime.h>

#include "NvInfer.h"
#include "common/bertCommon.h"
#include "common/common.cuh"
#include "common/serialize.hpp"
#include "cosLUPlugin.h"

using namespace nvinfer1;

namespace nvinfer1
{
namespace plugin
{

template <typename T, unsigned TPB>
__global__ void cosLUKernel(int n, const T* input, T* output, const T* a, const T* b)
{
    const int idx = blockIdx.x * TPB + threadIdx.x;

    if (idx < n)
    {
        const T in = input[idx];
        const T a_val = *(a);
        const T b_val = *(b);
        T sigmoid_val = 1 / (1 + __expf(-1*in)); 
        T cos_val = __cosf(b_val * in);
        output[idx] = sigmoid_val * (in + (a_val * cos_val));
    }
}

int computeCosLU(hipStream_t stream, int n, const float* input, float* output, const float* a, const float* b)
{
    constexpr int blockSize = 256; // number of threads, n = size of input
    const int gridSize = (n + blockSize - 1) / blockSize;
    cosLUKernel<float, blockSize><<<gridSize, blockSize, 0, stream>>>(n, input, output, a, b);

    PLUGIN_CHECK(hipPeekAtLastError());
    return 0;
}
} // namespace plugin
} // namespace nvinfer1
#endif // CUDA_VERSION >= 10010