#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <torch/script.h>

#if CUDA_VERSION >= 10010

#include <cstring>
#include <vector>

#include "NvInfer.h"
#include "common/bertCommon.h"
#include "common/common.cuh"
#include "common/serialize.hpp"
#include "cosLUPlugin.h"

using namespace nvinfer1;

namespace nvinfer1
{
namespace plugin
{
namespace bert
{

template <typename T, unsigned TPB>
__global__ void cosLUKernel(int n, const T* input, T* output, const T* a, const T* b)
{
    const int idx = blockIdx.x * TPB + threadIdx.x;

    if (idx < n)
    {
        const T in = input[idx];
        output[idx] = torch::sigmoid(in) * (in + (a * torch::cos(b*in)));;
    }
}

int computeCosLU(hipStream_t stream, int n, const float* input, float* output, const float* a, const float* b)
{
    constexpr int blockSize = 256;
    const int gridSize = (n + blockSize - 1) / blockSize;
    cosLUKernel<float, blockSize><<<gridSize, blockSize, 0, stream>>>(n, input, output, a, b);

    PLUGIN_CHECK(hipPeekAtLastError());
    return 0;
}

int computeCosLU(hipStream_t stream, int n, const half* input, half* output, const half* a, const half* b)
{
    constexpr int blockSize = 256;

    if (0 == (n & 1))
    {
        const int n2 = n / 2;

        const int gridSize = (n2 + blockSize - 1) / blockSize;
        const half2* input2 = reinterpret_cast<const half2*>(input);
        half2* output2 = reinterpret_cast<half2*>(output);
        const half2* a2 = reinterpret_cast<const half2*>(a);
        const half2* b2 = reinterpret_cast<const half2*>(b);
        cosLUKernel<half2, blockSize><<<gridSize, blockSize, 0, stream>>>(n2, input2, output2, a2, b2);
    }
    else
    {
        const int gridSize = (n + blockSize - 1) / blockSize;
        cosLUKernel<half, blockSize><<<gridSize, blockSize, 0, stream>>>(n, input, output, a, b);
    }

    PLUGIN_CHECK(hipPeekAtLastError());
    return 0;
}
} // namespace bert
} // namespace plugin
} // namespace nvinfer1
#endif // CUDA_VERSION >= 10010